
#include <hip/hip_runtime.h>
#include <iostream>
#include <cmath>

// nvcc cuda.cu -Xcompiler=-fPIC -g -gencode arch=compute_12,code=sm_12

__global__
void add(int n, float *x, float *y) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;
    for (int i = index; i < n; i += stride) {
        y[i] += x[i];
    }
}

__global__
void init(int n, float *x, float *y) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;
    for (int i = index; i < n; i += stride) {
        x[i] = i;
        y[i] = i;
    }
}

#define CHECK(statement) { \
    hipError_t res = (statement); \
    if (res != 0) { \
        std::cout << __PRETTY_FUNCTION__ << ":" << __LINE__ << ": "<< #statement << ": " << hipGetErrorString(res) << "\n"; \
        exit(1); \
    } \
}

int main(void) {
    int N = 1<<20;

    float *x;
    CHECK(hipMalloc(&x, N * sizeof(float)));

    float *y;
    CHECK(hipMalloc(&y, N * sizeof(float)));

    struct { float *x, *y; } host;
    CHECK(hipHostMalloc(&host.x, sizeof(float) * N, hipHostMallocDefault));
    CHECK(hipHostMalloc(&host.y, sizeof(float) * N, hipHostMallocDefault));

    int blockSize = 512;
    int numBlocks = (N + blockSize - 1) / blockSize;

    for (int i = 0; i < 1<<20; ++i) {
        init<<<numBlocks, blockSize>>>(N, x, y);
        CHECK(hipDeviceSynchronize());

        add<<<numBlocks, blockSize>>>(N, x, y);
        CHECK(hipDeviceSynchronize());
    }
    CHECK(hipMemcpy(host.x, x, sizeof(float) * N, hipMemcpyDeviceToHost));
    CHECK(hipMemcpy(host.y, y, sizeof(float) * N, hipMemcpyDeviceToHost));

    hipFree(x);
    hipFree(y);
    hipHostFree(host.x);
    hipHostFree(host.y);

    return 0;
}